#include "hip/hip_runtime.h"
/**
    This file is part of stixels. (https://github.com/dhernandez0/stixels).

    Copyright (c) 2016 Daniel Hernandez Juarez.

    stixels is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    stixels is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with stixels.  If not, see <http://www.gnu.org/licenses/>.

**/

#include "RoadEstimation.h"
#include "RoadEstimationKernels.cuh"
#include "util.hpp"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <vector>

class RoadEstimation::RoadEstimationImpl
{
public:
    RoadEstimationImpl();
    ~RoadEstimationImpl();
    void Initialize(const float camera_center_y, const float baseline, const float focal, const int rows,
        const int cols, const int max_dis);
    bool Compute(const pixel_t *im);
    void Finish();
    float GetCameraHeight() {return m_cameraHeight;};
    float GetPitch() {return m_pitch;};
    float GetSlope() {return m_slope;};
    int getHorizonPoint() {return m_horizonPoint;};
private:
    void ComputeCameraProperties(cv::Mat vDisp, const float rho, const float theta, float& horizonPoint,
        float& pitch, float& cameraHeight, float& slope) const;
    bool ComputeHough(uint8_t *vDisp, float& rho, float& theta, float& horizonPoint, float& pitch,
        float& cameraHeight, float& slope);

private: /*CUDA Host Pointer*/
    int m_rangeAngleX;          ///< Angle interval to discard horizontal planes
    int m_rangeAngleY;          ///< Angle interval to discard vertical planes
    int m_HoughAccumThr;        ///< Threshold of the min number of points to form a line
    float m_binThr;             ///< Threshold to binarize vDisparity histogram
    float m_maxPitch;			///< Angle elevation maximun of camera
    float m_minPitch;			///< Angle elevation minimum of camera
    float m_maxCameraHeight;    ///< Height maximun of camera
    float m_minCameraHeight;    ///< Height minimun of camera
    int m_max_dis;
    int m_rows;
    int m_cols;

    // Member objects
    float m_rho;                 ///< Line in polar (Distance from (0,0) to the line)
    float m_theta;               ///< Line in polar (Angle of the line with x axis)

    // Auxiliar variables
    int m_horizonPoint;          ///< Horizon point of v-disparity histogram
    float m_pitch;               ///< Camera pitch
    float m_cameraHeight;        ///< Camera height
    float m_cy;                  ///< Image center from stereo camera
    float m_b;                   ///< Stereo camera baseline
    float m_focal;               ///< Stereo camera focal length
    float m_slope;

private: /*CUDA Device Pointer*/
    pixel_t *d_disparity;
    int *d_vDisp;
    int *d_maximum;
    uint8_t *d_vDispBinary;
    uint8_t *m_vDisp;
};

RoadEstimation::RoadEstimationImpl::RoadEstimationImpl() {}

RoadEstimation::RoadEstimationImpl::~RoadEstimationImpl() {}

void RoadEstimation::RoadEstimationImpl::Initialize(const float camera_center_y, const float baseline, const float focal,
		const int rows, const int cols, const int max_dis) {
	// Get camera parameters
	m_cy = camera_center_y;
	m_b = baseline;
	m_focal = focal;

	// Default configuration
	m_rangeAngleX = 5;
	m_rangeAngleY = 5;
	m_HoughAccumThr = 25;
	m_binThr = 0.5f;
	m_maxPitch = 50;
	m_minPitch = -50;
	/*
	m_maxCameraHeight = -1.30f;
	m_minCameraHeight = -1.90f;
	 */
	m_maxCameraHeight = 1.90f;
	m_minCameraHeight = 1.30f;

	m_maxPitch = m_maxPitch*(float)CV_PI/180.0f;
	m_minPitch = m_minPitch*(float)CV_PI/180.0f;
	m_max_dis = max_dis;
	m_rows = rows;
	m_cols = cols;

	m_rho = 0;
	m_theta = 0;
	m_horizonPoint = 0;
	m_pitch = 0;
	m_cameraHeight = 0;

	m_vDisp = (uint8_t*) malloc(m_max_dis*m_rows*sizeof(uint8_t));

	CUDA_CHECK_RETURN(hipMalloc((void **)&d_disparity, m_cols*m_rows*sizeof(float)));
	CUDA_CHECK_RETURN(hipMalloc((void **)&d_vDisp, m_max_dis*m_rows*sizeof(int)));
	CUDA_CHECK_RETURN(hipMalloc((void **)&d_maximum, 1*sizeof(int)));
	CUDA_CHECK_RETURN(hipMalloc((void **)&d_vDispBinary, m_max_dis*m_rows*sizeof(uint8_t)));
}

void RoadEstimation::RoadEstimationImpl::Finish() {
	CUDA_CHECK_RETURN(hipFree(d_vDisp));
	CUDA_CHECK_RETURN(hipFree(d_disparity));
	CUDA_CHECK_RETURN(hipFree(d_maximum));
	CUDA_CHECK_RETURN(hipFree(d_vDispBinary));
	free(m_vDisp);
}

bool RoadEstimation::RoadEstimationImpl::Compute(const pixel_t *im) {
	bool ok = false;

	CUDA_CHECK_RETURN(hipMemset(d_maximum, 0, 1*sizeof(int)));
	CUDA_CHECK_RETURN(hipMemset(d_vDisp, 0, m_max_dis*m_rows*sizeof(int)));

	// Compute the vDisparity histogram
	CUDA_CHECK_RETURN(hipMemcpy(d_disparity, im, m_rows*m_cols*sizeof(pixel_t), hipMemcpyHostToDevice));
	ComputeHistogram<<<(m_rows*m_cols+256-1)/256, 256>>>(d_disparity, d_vDisp, m_rows, m_cols, m_max_dis);
	ComputeMaximum<<<(m_rows*m_max_dis+256-1)/256, 256>>>(d_vDisp, d_maximum, m_rows, m_max_dis);
	ComputeBinaryImage<<<(m_rows*m_max_dis+256-1)/256, 256>>>(d_vDisp, d_vDispBinary, d_maximum, m_binThr,
			m_rows, m_max_dis);

    // Compute the Hough transform
	float rho, theta, horizonPoint, pitch, cameraHeight, slope;
	if (ComputeHough(d_vDispBinary, rho, theta, horizonPoint, pitch, cameraHeight, slope)) {
		m_rho = rho;
		m_theta = theta;
		m_horizonPoint = (int) ceil(horizonPoint);
		m_pitch = pitch;
		m_cameraHeight = cameraHeight;
		m_slope = slope;
		ok = true;
	}

	return ok;
}

bool RoadEstimation::RoadEstimationImpl::ComputeHough(uint8_t *d_vDispBinary, float& rho, float& theta, float& horizonPoint,
		float& pitch, float& cameraHeight, float& slope) {
	// Compute the Hough transform
	std::vector<cv::Vec2f> lines;
	hipMemcpy(m_vDisp, d_vDispBinary, m_max_dis*m_rows*sizeof(uint8_t), hipMemcpyDeviceToHost);
	cv::Mat vDisp(m_rows, m_max_dis, CV_8UC1, m_vDisp);
	cv::HoughLines(vDisp, lines, 1.0, CV_PI/180, m_HoughAccumThr);

	// Get the best line from hough
	for (size_t i=0; i<lines.size(); i++) {
		// Get rho and theta
		rho = abs(lines[i][0]);
		theta = lines[i][1];

		// Compute camera position
		ComputeCameraProperties(vDisp, rho, theta, horizonPoint, pitch, cameraHeight, slope);

		//printf("%f (%f %f) %f (%f %f)\n", pitch, m_minPitch, m_maxPitch, cameraHeight, m_minCameraHeight, m_maxCameraHeight);
		//if (pitch>=m_minPitch && pitch<=m_maxPitch && cameraHeight>=m_minCameraHeight && cameraHeight<=m_maxCameraHeight) {
		if (pitch>=m_minPitch && pitch<=m_maxPitch) {
			return true;
		}
	}

	return false;
}

void RoadEstimation::RoadEstimationImpl::ComputeCameraProperties(cv::Mat vDisp, const float rho, const float theta,
		float& horizonPoint, float& pitch, float& cameraHeight, float& slope) const
{
	// Compute Horizon Line (2D)
	horizonPoint = rho/sinf(theta);

	// Compute pitch -> arctan((cy - y0Hough)/focal) It is negative because y axis is inverted
	pitch = -atanf((m_cy - horizonPoint)/(m_focal));

	// Compute the slope needed to compute the Camera height
	float last_row = (float)(vDisp.rows-1);
	float vDispDown = (rho-last_row*sinf(theta))/cosf(theta);
	slope = (0 - vDispDown)/(horizonPoint - last_row);

	// Compute the camera height -> baseline*cos(pitch)/slopeHough
	cameraHeight = m_b*cosf(pitch)/slope;
}

RoadEstimation::RoadEstimation() : 
    m_impl(new RoadEstimationImpl()) {}

RoadEstimation::~RoadEstimation() {}

void RoadEstimation::Initialize(const float camera_center_y, const float baseline, const float focal, const int rows,
    const int cols, const int max_dis) {
    m_impl->Initialize(camera_center_y, baseline, focal, rows, cols, max_dis);
}

bool RoadEstimation::Compute(const pixel_t *im) {
    return m_impl->Compute(im);
}

void RoadEstimation::Finish() {
    m_impl->Finish();
}

float RoadEstimation::GetCameraHeight() {
    return m_impl->GetCameraHeight();
}

float RoadEstimation::GetPitch() {
    return m_impl->GetPitch();
}

float RoadEstimation::GetSlope() {
    return m_impl->GetSlope();
}

int RoadEstimation::getHorizonPoint() {
    return m_impl->getHorizonPoint();
}
