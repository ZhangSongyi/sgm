/**
    This file is part of sgm. (https://github.com/dhernandez0/sgm).

    Copyright (c) 2016 Daniel Hernandez Juarez.

    sgm is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    sgm is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with sgm.  If not, see <http://www.gnu.org/licenses/>.

**/

#include "DisparityEstimation.h"
#include <stdint.h>
#include <opencv2/opencv.hpp>
#include "util.hpp"
#include "configuration.h"
#include "CenterSymmetricCensusKernel.cuh"
#include "HammingDistanceCostKernel.cuh"
#include "MedianFilterKernels.cuh"
#include "CostAggregationKernels.hpp"
#include "debug.h"

class DisparityEstimationImpl
{
public:
    DisparityEstimationImpl() {}
    ~DisparityEstimationImpl() {}
    void Initialize(const uint8_t p1, const uint8_t p2);
    cv::Mat Compute(cv::Mat left, cv::Mat right, float *elapsed_time_ms);
    void Finish();

private: /*CUDA Host Pointer*/
    uint8_t *h_disparity;
    uint8_t p1, p2;
    bool first_alloc;
    uint32_t cols, rows, size, size_cube_l;

private: /*CUDA Device Pointer*/
    hipStream_t stream1, stream2, stream3;//, stream4, stream5, stream6, stream7, stream8;
    uint8_t *d_im0;
    uint8_t *d_im1;
    cost_t *d_transform0;
    cost_t *d_transform1;
    uint8_t *d_cost;
    uint8_t *d_disparity;
    uint8_t *d_disparity_filtered_uchar;
    uint16_t *d_S;
    uint8_t *d_L0;
    uint8_t *d_L1;
    uint8_t *d_L2;
    uint8_t *d_L3;
    uint8_t *d_L4;
    uint8_t *d_L5;
    uint8_t *d_L6;
#if PATH_AGGREGATION == 8
    uint8_t *d_L7;
#endif

private:
    void malloc_memory();
    void free_memory();
};

void DisparityEstimationImpl::Initialize(const uint8_t p1, const uint8_t p2) {
    // We are not using shared memory, use L1
    //CUDA_CHECK_RETURN(hipDeviceSetCacheConfig(hipFuncCachePreferL1));
    //CUDA_CHECK_RETURN(hipDeviceSetCacheConfig(hipFuncCachePreferShared));

    // Create streams
    CUDA_CHECK_RETURN(hipStreamCreate(&stream1));
    CUDA_CHECK_RETURN(hipStreamCreate(&stream2));
    CUDA_CHECK_RETURN(hipStreamCreate(&stream3));
    first_alloc = true;
    this->p1 = p1;
    this->p2 = p2;
    rows = 0;
    cols = 0;
}

cv::Mat DisparityEstimationImpl::Compute(cv::Mat left, cv::Mat right, float *elapsed_time_ms) {
	if(cols != left.cols || rows != left.rows) {
		debug_log("WARNING: cols or rows are different");
		if(!first_alloc) {
			debug_log("Freeing memory");
			free_memory();
		}
		first_alloc = false;
		cols = left.cols;
		rows = left.rows;
		size = rows*cols;
		size_cube_l = size*MAX_DISPARITY;
        malloc_memory();
	}
	debug_log("Copying images to the GPU");
	CUDA_CHECK_RETURN(hipMemcpyAsync(d_im0, left.ptr<uint8_t>(), sizeof(uint8_t)*size, hipMemcpyHostToDevice, stream1));
	CUDA_CHECK_RETURN(hipMemcpyAsync(d_im1, right.ptr<uint8_t>(), sizeof(uint8_t)*size, hipMemcpyHostToDevice, stream1));

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	dim3 block_size;
	block_size.x = 32;
	block_size.y = 32;

	dim3 grid_size;
	grid_size.x = (cols+block_size.x-1) / block_size.x;
	grid_size.y = (rows+block_size.y-1) / block_size.y;

	debug_log("Calling CSCT");
	CenterSymmetricCensusKernelSM2<<<grid_size, block_size, 0, stream1>>>(d_im0, d_im1, d_transform0, d_transform1, rows, cols);

	// Hamming distance
	CUDA_CHECK_RETURN(hipStreamSynchronize(stream1));
	debug_log("Calling Hamming Distance");
	HammingDistanceCostKernel<<<rows, MAX_DISPARITY, 0, stream1>>>(d_transform0, d_transform1, d_cost, rows, cols);

	// Cost Aggregation
	const int PIXELS_PER_BLOCK = COSTAGG_BLOCKSIZE/WARP_SIZE;
	const int PIXELS_PER_BLOCK_HORIZ = COSTAGG_BLOCKSIZE_HORIZ/WARP_SIZE;

	debug_log("Calling Left to Right");
	CostAggregationKernelLeftToRight<<<(rows+PIXELS_PER_BLOCK_HORIZ-1)/PIXELS_PER_BLOCK_HORIZ, COSTAGG_BLOCKSIZE_HORIZ, 0, stream2>>>(d_cost, d_L0, p1, p2, rows, cols, d_transform0, d_transform1, d_disparity, d_L0, d_L1, d_L2, d_L3, d_L4, d_L5, d_L6);
	debug_log("Calling Right to Left");
	CostAggregationKernelRightToLeft<<<(rows+PIXELS_PER_BLOCK_HORIZ-1)/PIXELS_PER_BLOCK_HORIZ, COSTAGG_BLOCKSIZE_HORIZ, 0, stream3>>>(d_cost, d_L1, p1, p2, rows, cols, d_transform0, d_transform1, d_disparity, d_L0, d_L1, d_L2, d_L3, d_L4, d_L5, d_L6);
	debug_log("Calling Up to Down");
	CostAggregationKernelUpToDown<<<(cols+PIXELS_PER_BLOCK-1)/PIXELS_PER_BLOCK, COSTAGG_BLOCKSIZE, 0, stream1>>>(d_cost, d_L2, p1, p2, rows, cols, d_transform0, d_transform1, d_disparity, d_L0, d_L1, d_L2, d_L3, d_L4, d_L5, d_L6);
	CUDA_CHECK_RETURN(hipDeviceSynchronize());
	debug_log("Calling Down to Up");
	CostAggregationKernelDownToUp<<<(cols+PIXELS_PER_BLOCK-1)/PIXELS_PER_BLOCK, COSTAGG_BLOCKSIZE, 0, stream1>>>(d_cost, d_L3, p1, p2, rows, cols, d_transform0, d_transform1, d_disparity, d_L0, d_L1, d_L2, d_L3, d_L4, d_L5, d_L6);

#if PATH_AGGREGATION == 8
	CostAggregationKernelDiagonalDownUpLeftRight<<<(cols+PIXELS_PER_BLOCK-1)/PIXELS_PER_BLOCK, COSTAGG_BLOCKSIZE, 0, stream1>>>(d_cost, d_L4, p1, p2, rows, cols, d_transform0, d_transform1, d_disparity, d_L0, d_L1, d_L2, d_L3, d_L4, d_L5, d_L6);
	CostAggregationKernelDiagonalUpDownLeftRight<<<(cols+PIXELS_PER_BLOCK-1)/PIXELS_PER_BLOCK, COSTAGG_BLOCKSIZE, 0, stream1>>>(d_cost, d_L5, p1, p2, rows, cols, d_transform0, d_transform1, d_disparity, d_L0, d_L1, d_L2, d_L3, d_L4, d_L5, d_L6);

	CostAggregationKernelDiagonalDownUpRightLeft<<<(cols+PIXELS_PER_BLOCK-1)/PIXELS_PER_BLOCK, COSTAGG_BLOCKSIZE, 0, stream1>>>(d_cost, d_L6, p1, p2, rows, cols, d_transform0, d_transform1, d_disparity, d_L0, d_L1, d_L2, d_L3, d_L4, d_L5, d_L6);
	CostAggregationKernelDiagonalUpDownRightLeft<<<(cols+PIXELS_PER_BLOCK-1)/PIXELS_PER_BLOCK, COSTAGG_BLOCKSIZE, 0, stream1>>>(d_cost, d_L7, p1, p2, rows, cols, d_transform0, d_transform1, d_disparity, d_L0, d_L1, d_L2, d_L3, d_L4, d_L5, d_L6);
#endif
	debug_log("Calling Median Filter");
	MedianFilter3x3<<<(size+MAX_DISPARITY-1)/MAX_DISPARITY, MAX_DISPARITY, 0, stream1>>>(d_disparity, d_disparity_filtered_uchar, rows, cols);

	hipEventRecord(stop, 0);
	CUDA_CHECK_RETURN(hipDeviceSynchronize());
	hipEventElapsedTime(elapsed_time_ms, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	debug_log("Copying final disparity to CPU");
	CUDA_CHECK_RETURN(hipMemcpy(h_disparity, d_disparity_filtered_uchar, sizeof(uint8_t)*size, hipMemcpyDeviceToHost));

	cv::Mat disparity(rows, cols, CV_8UC1, h_disparity);
	return disparity;
}

void DisparityEstimationImpl::Finish() {
	if(!first_alloc) {
		free_memory();
		CUDA_CHECK_RETURN(hipStreamDestroy(stream1));
		CUDA_CHECK_RETURN(hipStreamDestroy(stream2));
		CUDA_CHECK_RETURN(hipStreamDestroy(stream3));
	}
}

void DisparityEstimationImpl::malloc_memory() {
    CUDA_CHECK_RETURN(hipMalloc((void **)&d_im0, sizeof(uint8_t)*size));
    CUDA_CHECK_RETURN(hipMalloc((void **)&d_im1, sizeof(uint8_t)*size));
    CUDA_CHECK_RETURN(hipMalloc((void **)&d_transform0, sizeof(cost_t)*size));
    CUDA_CHECK_RETURN(hipMalloc((void **)&d_transform1, sizeof(cost_t)*size));
    CUDA_CHECK_RETURN(hipMalloc((void **)&d_L0, sizeof(uint8_t)*size_cube_l));
    CUDA_CHECK_RETURN(hipMalloc((void **)&d_L1, sizeof(uint8_t)*size_cube_l));
    CUDA_CHECK_RETURN(hipMalloc((void **)&d_L2, sizeof(uint8_t)*size_cube_l));
    CUDA_CHECK_RETURN(hipMalloc((void **)&d_L3, sizeof(uint8_t)*size_cube_l));
#if PATH_AGGREGATION == 8
    CUDA_CHECK_RETURN(hipMalloc((void **)&d_L4, sizeof(uint8_t)*size_cube_l));
    CUDA_CHECK_RETURN(hipMalloc((void **)&d_L5, sizeof(uint8_t)*size_cube_l));
    CUDA_CHECK_RETURN(hipMalloc((void **)&d_L6, sizeof(uint8_t)*size_cube_l));
    CUDA_CHECK_RETURN(hipMalloc((void **)&d_L7, sizeof(uint8_t)*size_cube_l));
#endif
    CUDA_CHECK_RETURN(hipMalloc((void **)&d_cost, sizeof(uint8_t)*size_cube_l));
    CUDA_CHECK_RETURN(hipMalloc((void **)&d_disparity, sizeof(uint8_t)*size));
    CUDA_CHECK_RETURN(hipMalloc((void **)&d_disparity_filtered_uchar, sizeof(uint8_t)*size));
    h_disparity = new uint8_t[size];
}

void DisparityEstimationImpl::free_memory() {
	CUDA_CHECK_RETURN(hipFree(d_im0));
	CUDA_CHECK_RETURN(hipFree(d_im1));
	CUDA_CHECK_RETURN(hipFree(d_transform0));
	CUDA_CHECK_RETURN(hipFree(d_transform1));
	CUDA_CHECK_RETURN(hipFree(d_L0));
	CUDA_CHECK_RETURN(hipFree(d_L1));
	CUDA_CHECK_RETURN(hipFree(d_L2));
	CUDA_CHECK_RETURN(hipFree(d_L3));
#if PATH_AGGREGATION == 8
	CUDA_CHECK_RETURN(hipFree(d_L4));
	CUDA_CHECK_RETURN(hipFree(d_L5));
	CUDA_CHECK_RETURN(hipFree(d_L6));
	CUDA_CHECK_RETURN(hipFree(d_L7));
#endif
	CUDA_CHECK_RETURN(hipFree(d_disparity));
	CUDA_CHECK_RETURN(hipFree(d_disparity_filtered_uchar));
	CUDA_CHECK_RETURN(hipFree(d_cost));

	delete[] h_disparity;
}

DisparityEstimation::DisparityEstimation() : 
    m_impl(new DisparityEstimationImpl()) {}

DisparityEstimation::~DisparityEstimation() {}

void DisparityEstimation::Initialize(const uint8_t p1, const uint8_t p2) {
    m_impl->Initialize(p1, p2);
}

cv::Mat DisparityEstimation::Compute(cv::Mat left, cv::Mat right, float *elapsed_time_ms) {
    return m_impl->Compute(left, right, elapsed_time_ms);
}

void DisparityEstimation::Finish() {
    m_impl->Finish();
}
